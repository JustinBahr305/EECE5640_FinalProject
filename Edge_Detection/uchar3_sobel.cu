// sobel.cu
// Created by Justin Bahr on 3/24/2025.
// EECE 5640 - High Performance Computing
// Sobel Filter CUDA Kernel using uchar3

#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>

using namespace std;

// Sobel Kernels
__constant__ int SOBEL_X[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
};

__constant__ int SOBEL_Y[3][3] = {
    {-1, -2, -1},
    {0,  0,  0},
    {1,  2,  1}
};

// CUDA kernel to convert RGB to Grayscale
__global__ void rgbToGrayscale_uchar3(uchar3 *rgb, unsigned char *gray, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int idx = y * width + x;
        uchar3 pixel = rgb[idx];
        gray[idx] = (unsigned char)(0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z);
    }
}

// CUDA kernel for Sobel edge detection
__global__ void sobelFilter(unsigned char *input, unsigned char *output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
    {
        int Gx = 0, Gy = 0;

        // Apply Sobel operators
        for (int i = -1; i <= 1; i++)
        {
            for (int j = -1; j <= 1; j++)
            {
                int pixel = input[(y + i) * width + (x + j)];
                Gx += pixel * SOBEL_X[i + 1][j + 1];
                Gy += pixel * SOBEL_Y[i + 1][j + 1];
            }
        }

        // Compute gradient magnitude
        int magnitude = sqrtf(Gx * Gx + Gy * Gy);
        output[y * width + x] = (magnitude > 255) ? 255 : magnitude;
    }
}

// Function to process the image on GPU
void processImageCUDA(unsigned char *h_rgbData, unsigned char *h_outputData, int width, int height)
{
    size_t rgbSize = width * height * sizeof(uchar3);;
    size_t graySize = width * height;

    uchar3 *d_rgb;
    unsigned char *d_gray, *d_output;

    // Allocate memory on GPU
    hipMalloc((void **)&d_rgb, rgbSize);
    hipMalloc((void **)&d_gray, graySize);
    hipMalloc((void **)&d_output, graySize);

    // Copy RGB data to GPU
    hipMemcpy(d_rgb, (uchar3 *)h_rgbData, rgbSize, hipMemcpyHostToDevice);

    // Define CUDA grid/block sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Convert to grayscale
    rgbToGrayscale_uchar3<<<gridSize, blockSize>>>(d_rgb, d_gray, width, height);
    hipDeviceSynchronize();

    // Apply Sobel filter
    sobelFilter<<<gridSize, blockSize>>>(d_gray, d_output, width, height);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_outputData, d_output, graySize, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_rgb);
    hipFree(d_gray);
    hipFree(d_output);
}